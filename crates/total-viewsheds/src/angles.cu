#include "hip/hip_runtime.h"
#include <cub/block/block_scan.cuh>

typedef struct {
    unsigned int angles;
    unsigned int total_bands;
    unsigned int max_los_as_points;
    unsigned int dem_width;
    unsigned int tvs_width;
    float observer_height;
} calculation_constants;

#define EARTH_RADIUS_SQUARED 12742000.0

#define TOTAL_BANDS 72000000
#define TVS_WIDTH 6000
#define MAX_LOS_POINTS 6000

#define BLOCK_SIZE 6

#define TAN_ONE_RAD 0.0174533

#define ull unsigned long long


extern "C" __global__ void angle_kernel(
    // Constants for the calculations.
    calculation_constants constants,
    // Every single DEM point's elevation.
    const float* __restrict__ elevations,
    // Every single DEM point's distance from the sector-orthogonal axis.
    const float* __restrict__ distances,
    // Deltas used to build a band. Deltas are always the same for both front and back bands.
    //
    // Deltas are simply the numerical difference between DEM IDs in a band of
    // sight. DEM IDs are certainly different for every point in a DEM, but
    // conveniently the _differences_ between DEM IDs are identical. With the only
    // caveat that back-facing bands have opposite magnitudes. It should be stressed
    // that this feature of band data is a huge benefit to the space-requirements
    // (and thus speed) of the algorithm.
    // TODO: Explore:
    //         * storing the band deltas in local memory.
    //         * compressing repeating delta values -- is simple addition faster than
    //           memory accesses?
    const unsigned int* __restrict__ delta_pos,
    const unsigned int* __restrict__ delta_neg,
    float* result,
    int offset
) {
    // line_num tells us what (kernel_id, angle) we are at
    ull line_num = offset + blockIdx.x;

//     printf("line_num: %d\n", line_num);

    ull tvs_id = line_num % TOTAL_BANDS;
    ull angle = line_num / TOTAL_BANDS;

    // determine whether we are forwards or backwards facing
    // TODO: this doesn't seem to "fall out" of the implementation
    ull half_total_bands = TOTAL_BANDS/2;

    bool forward = tvs_id > half_total_bands;
    if (forward) {
        tvs_id -= half_total_bands;
    }

    ull pov_x = (tvs_id % TVS_WIDTH) + MAX_LOS_POINTS;
    ull pov_y = (tvs_id / TVS_WIDTH) + MAX_LOS_POINTS;

    // get the dem id for our pov which is where we start our calculation
    ull pov_id = (pov_x * constants.dem_width) + pov_y;

    // calculate he height
    const float pov_elevation = elevations[pov_id] + constants.observer_height;


    float angle_buf[BLOCK_SIZE];
    float prefix_max[BLOCK_SIZE];
    float distance[BLOCK_SIZE];

    int delta_index_start = (angle*MAX_LOS_POINTS) + threadIdx.x*BLOCK_SIZE;

    #pragma unroll
    for (int i = 0; i < BLOCK_SIZE; i++) {
        int delta = forward ? delta_pos[delta_index_start+i]
                            : -delta_neg[delta_index_start+i];

        distance[i] = distances[delta_index_start + i];

        ull dem_id = pov_id + delta;

        float elevation_delta = elevations[dem_id] - pov_elevation;
        angle_buf[i] = elevation_delta / distance[i];
    }

    __syncthreads();

    using BlockScan = hipcub::BlockScan<float, 1000>;
    __shared__ typename BlockScan::TempStorage temp_storage;

    BlockScan(temp_storage)
        .InclusiveScan(angle_buf, prefix_max, cuda::maximum<>{});


    float sum = 0.0;

    #pragma unroll
    for (int i = 0; i < BLOCK_SIZE; i++) {
        if (angle_buf[i] >= prefix_max[i]) {
            sum += distance[i] * TAN_ONE_RAD;
        }
    }


    if (sum > 0.0) {
        atomicAdd(&result[tvs_id], sum);
    }
}
