
#include <hip/hip_runtime.h>


typedef struct {
    unsigned int total_bands;
    unsigned int max_los_as_points;
    unsigned int dem_width;
    unsigned int tvs_width;
    float observer_height;
    unsigned int reserved_rings;
} calculation_constants;

#define EARTH_RADIUS_SQUARED 12742000.0

extern "C" __global__ void angle_kernel(
    // Constants for the calculations.
    calculation_constants constants,
    // Every single DEM point's elevation.
    const float* __restrict__ elevations,
    // Every single DEM point's distance from the sector-orthogonal axis.
    const float* __restrict__ distances,
    // Deltas used to build a band. Deltas are always the same for both front and back bands.
    //
    // Deltas are simply the numerical difference between DEM IDs in a band of
    // sight. DEM IDs are certainly different for every point in a DEM, but
    // conveniently the _differences_ between DEM IDs are identical. With the only
    // caveat that back-facing bands have opposite magnitudes. It should be stressed
    // that this feature of band data is a huge benefit to the space-requirements
    // (and thus speed) of the algorithm.
    // TODO: Explore:
    //         * storing the band deltas in local memory.
    //         * compressing repeating delta values -- is simple addition faster than
    //           memory accesses?
    const unsigned int* __restrict__ delta_pos,
    const unsigned int* __restrict__ delta_neg,
    float* result
) {
    int half_total_bands = constants.total_bands/2;

    int tvs_id = ((blockDim.x*blockIdx.x)+threadIdx.x) % half_total_bands;

    bool forward = tvs_id < half_total_bands;

    int pov_x = (tvs_id % constants.tvs_width) + constants.max_los_as_points;
    int pov_y = (tvs_id / constants.tvs_width) + constants.max_los_as_points;

    // get the dem id for our pov which is where we start our calculation
    int pov_id = (pov_x * constants.dem_width) + pov_y;

    // calculate he height
    float pov_elevation = elevations[pov_id] + constants.observer_height;
    float pov_distance = distances[pov_id];

    int delta = forward ? delta_pos[threadIdx.y]
                        : -delta_neg[threadIdx.y];

    int dem_id = pov_id + delta;

    float elevation_delta = elevations[dem_id] - pov_elevation;
    float distance_delta = fabs(distances[dem_id] - pov_distance);

    int absolute_pov_idx = blockDim.x * blockIdx.x * blockDim.y;
    int index = blockDim.x * blockIdx.x * blockDim.y
        + threadIdx.y * blockDim.x + threadIdx.x;

    result[index] = (elevation_delta/distance_delta);
}
